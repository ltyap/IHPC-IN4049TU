// the subroutine for GPU code can be found in several separated text file from the Brightspace. 
// You can add these subroutines to this main code.
////////////////////////////////////////////


#include <stdio.h>
#include <math.h>
#include <stdlib.h>
#include <string.h>
#include <time.h>
#include "hip/hip_runtime.h"


//const int BLOCK_SIZE = 32;  // number of threads per block

// Input Array Variables
float* h_MatA = NULL;
float* d_MatA = NULL;

// Output Array
float* h_VecV = NULL;
float* d_VecV = NULL;
float* h_VecW = NULL;
float* d_VecW = NULL;
float* h_NormW = NULL;
float* d_NormW = NULL;



// Variables to change
int GlobalSize = 10000;         // this is the dimension of the matrix, GlobalSize*GlobalSize
int BlockSize = 32;            // number of threads in each block
const float EPS = 0.0000001;    // tolerence of the error
int max_iteration = 200;       // the maximum iteration steps

// Functions
void Cleanup(void);
void InitOne(float*, int);
void UploadArray(float*, int);
float CPUReduce(float*, int);
void  Arguments(int, char**);
void checkCardVersion(void);

// Kernels
__global__ void Av_Product(float* g_MatA, float* g_VecV, float* g_VecW, int N);
__global__ void FindNormW(float* g_VecW, float* g_NormW, int N);
__global__ void NormalizeW(float* g_VecV,float* g_VecW, int N);
__global__ void ComputeLamda( float* g_VecV,float* g_VecW, float* g_Lamda,int N);


void CPU_AvProduct()
{
	int N = GlobalSize;
	int matIndex =0;
    for(int i=0;i<N;i++)
	{
		h_VecW[i] = 0;
		for(int j=0;j<N;j++)
		{
			matIndex = i*N + j;
			h_VecW[i] += h_MatA[matIndex] * h_VecV[j];
			
		}
	}
}

void CPU_NormalizeW()
{
	int N = GlobalSize;
	float normW=0;
	for(int i=0;i<N;i++)
		normW += h_VecW[i] * h_VecW[i];
	
	normW = sqrt(normW);
	for(int i=0;i<N;i++)
		h_VecV[i] = h_VecW[i]/normW;
}

float CPU_ComputeLamda()
{
	int N = GlobalSize;
	float lamda =0;
	for(int i=0;i<N;i++)
		lamda += h_VecV[i] * h_VecW[i];
	
	return lamda;
}

void RunCPUPowerMethod()
{
	printf("*************************************\n");
	float oldLambda =0;
	float lamda=0;
	
	//AvProduct
	CPU_AvProduct();
	
	//power loop
	for (int i=0;i<max_iteration;i++)
	{
		CPU_NormalizeW();
		CPU_AvProduct();
		lamda= CPU_ComputeLamda();
		printf("CPU lamda at %d: %f \n", i, lamda);
		// If residual is lass than epsilon break
		if(abs(oldLambda - lamda) < EPS)
			break;
		oldLambda = lamda;	
	
	}
	printf("*************************************\n");
	
}

/*****************************************************************************
This function finds the product of Matrix A and vector V
******************************************************************************
// parallelization method for the Matrix-vector multiplication as follows: 

// each thread handle a multiplication of each row of Matrix A and vector V;

// The share memory is limited for a block, instead of reading an entire row of matrix A or vector V from global memory to share memory, 
// a square submatrix of A is shared by a block, the size of square submatrix is BLOCK_SIZE*BLOCK_SIZE; Thus, a for-loop is used to
// handle a multiplication of each row of Matrix A and vector V step by step. In each step, two subvectors with size BLOCK_SIZE is multiplied.
***************************************************************************************************************************************************/
__global__ void Av_Product(float* g_MatA, float* g_VecV, float* g_VecW, int N)
{ 
    unsigned int globalid = blockIdx.x * blockDim.x + threadIdx.x;
    float Csub = 0;

    for (int k = 0; k < N; ++k) 
    {
        Csub += g_MatA[globalid*N+k] * g_VecV[k];
    }
    __syncthreads();
    g_VecW[globalid] = Csub;

}

__global__ void ComputeLamda(float* g_VecV, float* g_VecW, float* g_Lamda, int N)
{
    unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;

   // atomic operations:
    atomicAdd(g_Lamda, g_VecV[globalid] * g_VecW[globalid]);    
}

/****************************************************
Normalizes vector W : W/norm(W)
****************************************************/
__global__ void FindNormW(float* g_VecW, float* g_NormW, int N)
{ 
   unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;

   // atomic operations:
   atomicAdd(g_NormW, g_VecW[globalid]*g_VecW[globalid]);
}

__global__ void NormalizeW(float* g_VecW, float* g_NormW, float* g_VecV, int N)
{
    unsigned int globalid = blockIdx.x*blockDim.x + threadIdx.x;
    g_VecV[globalid] = g_VecW[globalid]/g_NormW[0];
    __syncthreads();
}

// Host code
int main(int argc, char** argv)
{
    struct timespec t_start,t_end;
    double runtime;
    Arguments(argc, argv);
		
    int N = GlobalSize;
    printf("Matrix size %d X %d \n", N, N);
    size_t vec_size = N * sizeof(float);
    size_t mat_size = N * N * sizeof(float);
    size_t norm_size = sizeof(float);

  
    // Allocate normalized value in host memory
    h_NormW = (float*)malloc(norm_size);
    // Allocate input matrix in host memory
    h_MatA = (float*)malloc(mat_size);
    // Allocate initial vector V in host memory
    h_VecV = (float*)malloc(vec_size);
    // Allocate W vector for computations
    h_VecW = (float*)malloc(vec_size);
    
    


    // Initialize input matrix
    UploadArray(h_MatA, N);
    InitOne(h_VecV,N);

    printf("Power method in CPU starts\n");	   
    clock_gettime(CLOCK_REALTIME,&t_start);
    RunCPUPowerMethod();   // the lamda is already solved here
    clock_gettime(CLOCK_REALTIME,&t_end);
    runtime = (t_end.tv_sec - t_start.tv_sec) + 1e-9*(t_end.tv_nsec - t_start.tv_nsec);
    printf("CPU: run time = %f secs.\n",runtime);
    printf("Power method in CPU is finished\n");
    
    
    /////////////////////////////////////////////////
    // This is the starting points of GPU
    printf("Power method in GPU starts(global mem)\n");
    checkCardVersion();
    int i;
    // Initialize input matrix
    InitOne(h_VecV,N);
    
    clock_gettime(CLOCK_REALTIME,&t_start);  // Here I start to count

    // Set the kernel arguments
    int threadsPerBlock = BlockSize;   
   // int sharedMemSize = threadsPerBlock * threadsPerBlock * sizeof(float); // in per block, the memory is shared   
    int blocksPerGrid = (N + threadsPerBlock - 1) / threadsPerBlock;

    // Allocate matrix and vectors in device memory
    hipMalloc((void**)&d_MatA, mat_size); 
    hipMalloc((void**)&d_VecV, vec_size); 
    hipMalloc((void**)&d_VecW, vec_size); // This vector is only used by the device
    hipMalloc((void**)&d_NormW, norm_size); 

   //Power method loops
    float oldLambda =0;
   
    //Copy from host memory to device memory
    hipMemcpy(d_MatA, h_MatA, mat_size, hipMemcpyHostToDevice);
    hipMemcpy(d_VecV, h_VecV, vec_size, hipMemcpyHostToDevice);
    // cutilCheckError(cutStopTimer(timer_mem));
	  
   //initial w-vector 
    Av_Product<<<blocksPerGrid, threadsPerBlock>>>(d_MatA, d_VecV, d_VecW, N);
    hipDeviceSynchronize(); //Needed, kind of barrier to sychronize all threads
	
    // This part is the main code of the iteration process for the Power Method in GPU. 
    // Please finish this part based on the given code. Do not forget the command line 
    // cudaThreadSynchronize() after calling the function every time in CUDA to synchoronize the threads
    ////////////////////////////////////////////
    //   ///      //        //            //          //            //        //

   	for (i=0;i<max_iteration;i++)
	{     
        
        h_NormW[0] = 0;
        hipMemcpy(d_NormW, h_NormW, norm_size, hipMemcpyHostToDevice);
        
        FindNormW<<<blocksPerGrid, threadsPerBlock>>>(d_VecW, d_NormW, N);
        hipDeviceSynchronize(); 
       
        //need to transfer from device to host ??
        hipMemcpy(h_NormW, d_NormW, norm_size, hipMemcpyDeviceToHost);
        
        h_NormW[0]=sqrt(h_NormW[0]);
        
        //transfer back to device ??
        hipMemcpy(d_NormW, h_NormW, norm_size, hipMemcpyHostToDevice);
       
        NormalizeW<<<blocksPerGrid, threadsPerBlock>>>(d_VecW, d_NormW, d_VecV, N);
        hipDeviceSynchronize(); 
        
        Av_Product<<<blocksPerGrid, threadsPerBlock>>>(d_MatA, d_VecV, d_VecW, N);
        hipDeviceSynchronize(); 
        

        h_NormW[0] = 0;
        hipMemcpy(d_NormW, h_NormW, norm_size, hipMemcpyHostToDevice);
        
        ComputeLamda<<<blocksPerGrid, threadsPerBlock>>>(d_VecV, d_VecW, d_NormW, N);//???
        hipDeviceSynchronize(); 

		//transfer d_NormW to h_NormW via cudaMemcpy
        hipMemcpy(h_NormW, d_NormW, norm_size, hipMemcpyDeviceToHost);

        printf("GPU lamda at %d: %f \n", i, h_NormW[0]);
		// If residual is less than epsilon break
		if(abs(oldLambda - h_NormW[0]) < EPS)
			break;
		oldLambda = h_NormW[0];//
	} 
    
    clock_gettime(CLOCK_REALTIME,&t_end);
    runtime = (t_end.tv_sec - t_start.tv_sec) + 1e-9*(t_end.tv_nsec - t_start.tv_nsec);
    printf("GPU: run time = %f secs.\n",runtime);
    printf("GPU: run time per iteration = %f secs.\n",runtime/(i+1));
    // printf("Overall CPU Execution Time: %f (ms) \n", cutGetTimerValue(timer_CPU));
    // printf("Overall CPU Execution Time: %f (ms) \n", cutGetTimerValue(timer_CPU));

    Cleanup();
}

void Cleanup(void)
{
    // Free device memory
    if (d_MatA)
        hipFree(d_MatA);
    if (d_VecV)
        hipFree(d_VecV);
    if (d_VecW)
        hipFree(d_VecW);
	  if (d_NormW)
		    hipFree(d_NormW);
		
    // Free host memory
    if (h_MatA)
        free(h_MatA);
    if (h_VecV)
        free(h_VecV);
    if (h_VecW)
        free(h_VecW);
    if (h_NormW)
        free(h_NormW);
    
    exit(0);
}

// Allocates an array with zero value.
void InitOne(float* data, int n)
{
    for (int i = 0; i < n; i++)
        data[i] = 0;
	data[0]=1;
}

void UploadArray(float* data, int n)
{
   int total = n*n;
   int value=1;
    for (int i = 0; i < total; i++)
    {
    	data[i] = (int) (rand() % (int)(101));//1;//value;
	    value ++; if(value>n) value =1;
      // data[i] = 1;
    }
}

// Obtain program arguments
void Arguments(int argc, char** argv)
{
    for (int i = 0; i < argc; ++i) 
    {
        if (strcmp(argv[i], "--size") == 0 || strcmp(argv[i], "-size") == 0)
        {
            GlobalSize = atoi(argv[i+1]);
		    i = i + 1;
        }
        if (strcmp(argv[i], "--max_iteration") == 0 || strcmp(argv[i], "-max_iteration") == 0)
        {
            max_iteration = atoi(argv[i+1]);
		    i = i + 1;
        }
    }
}


void checkCardVersion()
{
   hipDeviceProp_t prop;
   
   hipGetDeviceProperties(&prop, 0);
   
   printf("This GPU has major architecture %d, minor %d \n",prop.major,prop.minor);
   if(prop.major < 2)
   {
      fprintf(stderr,"Need compute capability 2 or higher.\n");
      exit(1);
   }
}
